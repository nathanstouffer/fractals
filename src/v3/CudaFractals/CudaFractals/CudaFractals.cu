#include "hip/hip_runtime.h"
// CudaFractals.cpp : This file contains the 'main' function. Program execution begins and ends there.
//

#include <iostream>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>

// macro to catch cuda failures
#define CUDA_ERROR(x) \
{ \
    hipError_t cudaErrorCode = ##x; \
    if (cudaErrorCode != hipSuccess) \
    { \
        std::cout << "cuda function failed on line " << __LINE__ << " of " << __FILE__ << std::endl; \
        std::cout << "error: " << hipGetErrorString(cudaErrorCode) << std::endl; \
    } \
}

__global__ void dAddInts(int* a, int* b, int* sum)
{
    sum[0] = a[0] + b[0];
    a[0] = 4;
    b[0] = 7;
}

int main()
{
    int hA = 5;
    int hB = 3;
    int hSum;

    int* dA;
    int* dB;
    int* dSum;

    // allocate bytes on the gpu
    CUDA_ERROR(hipMalloc((void**)&dA, sizeof(int)));
    CUDA_ERROR(hipMalloc((void**)&dB, sizeof(int)));
    CUDA_ERROR(hipMalloc((void**)&dSum, sizeof(int)));

    // copy cpu bytes over to gpu
    CUDA_ERROR(hipMemcpy(dA, &hA, sizeof(int), hipMemcpyHostToDevice));
    CUDA_ERROR(hipMemcpy(dB, &hB, sizeof(int), hipMemcpyHostToDevice));

    // add the ints on the gpu
    dAddInts<<<1, 1 >>>(dA, dB, dSum);

    // copy the memory from the gpu to the cpu
    CUDA_ERROR(hipMemcpy(&hSum, dSum, sizeof(int), hipMemcpyDeviceToHost));
    
    // temp for debugging
    int tmp0;
    int tmp1;
    CUDA_ERROR(hipMemcpy(&tmp0, dA, sizeof(int), hipMemcpyDeviceToHost));
    CUDA_ERROR(hipMemcpy(&tmp1, dB, sizeof(int), hipMemcpyDeviceToHost));
    std::cout << tmp0 << std::endl;
    std::cout << tmp1 << std::endl;

    // free gpu memory
    CUDA_ERROR(hipFree(dA));
    CUDA_ERROR(hipFree(dB));
    CUDA_ERROR(hipFree(dSum));

    // print the output
    std::cout << "5 + 3 = " << hSum << std::endl;

    return 0;
}
